#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"
#include "matrixmul_kernel.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
    // M is j by k
    // N is k by l
    // P is j by l

    unsigned int j = M.height; // or P.height since they will be the same 
    unsigned int k = N.height; // or M.width since they need to be the same
    unsigned int l = P.width;  // or N.width since they will be the same

    __shared__ float ms[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ns[TILE_WIDTH][TILE_WIDTH];

    unsigned int bx = blockIdx.x, by = blockIdx.y;
    unsigned int tx = threadIdx.x, ty = threadIdx.y;

    unsigned int row = by*TILE_WIDTH + ty;
    unsigned int col = bx*TILE_WIDTH + tx;

    float pval = 0;

    float* me = M.elements;
    float* ne = N.elements;

    for (int phase = 0; phase < max(ceil(k/(float)TILE_WIDTH), ceil(l/(float)TILE_WIDTH)); phase++){

        // need to make sure we are still within the bounds of the matrix
        if ((row < j) && (phase*TILE_WIDTH+tx) < k){
            ms[ty][tx] = me[row*k + (phase*TILE_WIDTH + tx)];
        }
        else {
            ms[ty][tx] = 0.0f;
        }
        if ((phase*TILE_WIDTH+ty) < k && (col < l)){
            ns[ty][tx] = ne[(phase*TILE_WIDTH + ty)*l + col];
        }
        else {
            ns[ty][tx] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++){
            pval += ms[ty][i] * ns[i][tx];
        }
        
        __syncthreads();
    }
    if ((row < P.height) && (col < P.width)){
        P.elements[row*l + col] = pval;
    }
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
